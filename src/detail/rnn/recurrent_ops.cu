#include "hip/hip_runtime.h"

// Persistent RNN Includes
#include <prnn/detail/rnn/recurrent_ops.h>

#include <prnn/detail/matrix/matrix_view.h>
#include <prnn/detail/matrix/matrix_operations.h>
#include <prnn/detail/matrix/blas_operations.h>
#include <prnn/detail/matrix/operation.h>

#include <prnn/detail/parallel/hip/hip_runtime.h>

#include <prnn/detail/util/metaprogramming.h>
#include <prnn/detail/util/logger.h>

#include <prnn/detail/rnn/recurrent_ops_config.h>
#include <prnn/detail/rnn/recurrent_ops_handle.h>
#include <prnn/detail/rnn/recurrent_ops_kernels.h>

namespace prnn
{

namespace rnn
{

namespace detail
{

class TileSizeSelector
{
public:
    TileSizeSelector(size_t major, size_t minor, size_t smCount,
        const matrix::Precision& precision)
    : streamingMultiprocessorVersionMajor(major),
      streamingMultiprocessorVersionMinor(minor),
      streamingMultiprocessorCount(smCount),
      precision(precision)
    {

    }

public:
    size_t getMaximumSize() const
    {
        if(streamingMultiprocessorVersionMajor == 6 && streamingMultiprocessorCount >= 60)
        {
            if(precision == matrix::HalfPrecision())
            {
                return 2720;
            }
            else
            {
                return 1820;
            }
        }
        else if(streamingMultiprocessorVersionMajor == 5 && streamingMultiprocessorCount >= 24)
        {
            return 1088;
        }
        else
        {
            return 224;
        }
    }

public:
    size_t streamingMultiprocessorVersionMajor;
    size_t streamingMultiprocessorVersionMinor;

    size_t streamingMultiprocessorCount;

public:
    matrix::Precision precision;

};

void getGPUMajorAndMinorVersion(int& major, int& minor, int& smCount)
{
    if(prnn::parallel::isCudaEnabled())
    {
        prnn::parallel::CudaRuntimeLibrary::hipDeviceGetAttribute(&major,
            prnn::parallel::CudaRuntimeLibrary::hipDeviceAttributeComputeCapabilityMajor, 0);
        prnn::parallel::CudaRuntimeLibrary::hipDeviceGetAttribute(&minor,
            prnn::parallel::CudaRuntimeLibrary::hipDeviceAttributeComputeCapabilityMajor, 0);
        prnn::parallel::CudaRuntimeLibrary::hipDeviceGetAttribute(&smCount,
            prnn::parallel::CudaRuntimeLibrary::hipDeviceAttributeMultiprocessorCount, 0);
    }
}

} // namespace detail

size_t getMaximumSizeRNNForThisGPU(const matrix::Precision& precision)
{
    int major   = 0;
    int minor   = 0;
    int smCount = 0;

    detail::getGPUMajorAndMinorVersion(major, minor, smCount);

    return detail::TileSizeSelector(major, minor, smCount, precision).getMaximumSize();
}

namespace detail
{

template <typename ArchitectureConfig>
static index_t* getSynchronizerScratch(typename ArchitectureConfig::RealType* scratch,
    const ArchitectureConfig& archParameters)
{
    size_t totalSize = archParameters.activations_per_grid() *
        archParameters.handle.miniBatchSize *
        archParameters.handle.timesteps;

    return reinterpret_cast<index_t*>(scratch + totalSize);
}

template <typename ActivationFunction, typename ArchitectureConfig>
void dispatchForwardPropRecurrent(typename ArchitectureConfig::RealType* activations,
    const typename ArchitectureConfig::RealType* weights,
    typename ArchitectureConfig::RealType* scratch, const ArchitectureConfig& archParameters)
{
    typedef typename ArchitectureConfig::RealType RealType;

    size_t activationCount = archParameters.handle.layerSize;
    size_t miniBatchSize   = archParameters.handle.miniBatchSize;
    size_t timesteps       = archParameters.handle.timesteps;

    util::log("RecurrentOperations") << "Launch forward propagation with "
        << archParameters.block_count() << " blocks ("
        << archParameters.thread_count() << " threads), each handling "
        << archParameters.activations_per_block() << " activations out of "
        << activationCount << " total, mini batch size " << miniBatchSize << ", timesteps "
        << timesteps << ".\n";

    hipStream_t stream = reinterpret_cast<hipStream_t>(archParameters.handle.stream);

    Synchronizer synchronizer(archParameters.block_count(), stream,
        getSynchronizerScratch(scratch, archParameters));

    typedef typename ArchitectureConfig::TileParameters TileConfig;

    typedef RecurrentConfig<RealType, ActivationFunction, TileConfig> Config;

    Config config(archParameters.handle);

    while(synchronizer.not_finished()) {
        PersistentEngineParameters<Config> parameters(config, weights, activations,
            scratch, archParameters.handle.skipConnectionScale, synchronizer);

        forward_prop_recurrent_kernel<<<archParameters.blocks(),
            archParameters.threads(), 0, stream>>>(parameters);

        synchronizer.check_for_failure();

        if (synchronizer.not_finished()) {
            util::log("RecurrentOperations") << " forward prop launch failed, restarting at phase "
                << synchronizer.get_current_phase() << ".\n";
            synchronizer.reset_failed_flag();
        }
    }

}

template<RecurrentLayerDirection direction, typename T, size_t sms, size_t smMajor>
class TileSelector
{
public:
    typedef TileConfig<sms, 224, 224, 224, 224, 14, 14, direction> TileSize;

};

template<RecurrentLayerDirection direction, typename T>
class TileSelector<direction, T, 60, 6>
{
public:
    typedef TileConfig<60, 1820, 1820, 96, 96, 12, 12, direction> TileSize;
};

template<RecurrentLayerDirection direction>
class TileSelector<direction, float16, 60, 6>
{
public:
    typedef TileConfig<60, 2720, 2720, 352, 352, 22, 22, direction> TileSize;
};

template<RecurrentLayerDirection direction, typename T>
class TileSelector<direction, T, 24, 5>
{
public:
    typedef TileConfig<24, 1088, 1088, 224, 224, 14, 14, direction> TileSize;
};

template <typename ActivationFunction, typename T, RecurrentLayerDirection direction>
void forwardPropRecurrent(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<T>& precision)
{
    typedef typename T::type RealType;

    const RealType* weightsData    = weights.data<RealType>();
          RealType* activationData = activations.data<RealType>();
          RealType* scratchData    = scratch.data<RealType>();

    int major = 0;
    int minor = 0;
    int smCount = 0;

    getGPUMajorAndMinorVersion(major, minor, smCount);

    if(major == 6 && smCount >= 60)
    {
        typedef typename TileSelector<direction, RealType, 60, 6>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchForwardPropRecurrent<ActivationFunction, ArchParams>(activationData, weightsData,
            scratchData, architectureConfig);
    }
    else if(major == 5 && smCount >= 24)
    {
        typedef typename TileSelector<direction, RealType, 24, 5>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchForwardPropRecurrent<ActivationFunction, ArchParams>(activationData, weightsData,
            scratchData, architectureConfig);
    }
    else
    {
        typedef typename TileSelector<direction, RealType, 1, 0>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchForwardPropRecurrent<ActivationFunction, ArchParams>(activationData, weightsData,
            scratchData, architectureConfig);
    }
}

template <typename ActivationFunction, typename T>
void forwardPropRecurrentOverPrecisions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<T>& precision)
{
    typedef T PossiblePrecision;

    assert(PossiblePrecision() == activations.precision());

    if(handle.direction == prnn::RECURRENT_REVERSE)
    {
        forwardPropRecurrent<ActivationFunction, PossiblePrecision, prnn::RECURRENT_REVERSE>(
            activations, weights, scratch, handle, precision);
    }
    else
    {
        forwardPropRecurrent<ActivationFunction, PossiblePrecision, prnn::RECURRENT_FORWARD>(
            activations, weights, scratch, handle, precision);
    }
}

template<typename ActivationFunction, typename Precisions>
void forwardPropRecurrentOverPrecisions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const Precisions& precisions)
{
    typedef typename std::tuple_element<0, Precisions>::type PossiblePrecision;

    if(activations.precision() == PossiblePrecision())
    {
        forwardPropRecurrentOverPrecisions<ActivationFunction>(
            activations, weights, scratch, handle, std::tuple<PossiblePrecision>());
    }
    else
    {
        typedef typename util::RemoveFirstType<Precisions>::type RemainingPrecisions;

        forwardPropRecurrentOverPrecisions<ActivationFunction>(activations, weights,
            scratch, handle, RemainingPrecisions());
    }
}

template <typename ActivationFunction>
void forwardPropRecurrentOverPrecisions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    forwardPropRecurrentOverPrecisions<ActivationFunction>(activations, weights, scratch,
        handle, prnn::matrix::AllPrecisions());
}

template<typename ActivationFunction>
void forwardPropRecurrentOverActivationFunctions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<ActivationFunction>& activationFunction)
{
    assert(ActivationFunction() == *handle.activationFunction.forwardOperation);

    forwardPropRecurrentOverPrecisions<ActivationFunction>(activations, weights, scratch, handle);
}

template<typename Functions>
void forwardPropRecurrentOverActivationFunctions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const Functions& functions)
{
    typedef typename std::tuple_element<0, Functions>::type PossibleFunction;

    if(*handle.activationFunction.forwardOperation == PossibleFunction())
    {
        forwardPropRecurrentOverActivationFunctions(activations, weights,
            scratch, handle, std::tuple<PossibleFunction>());
    }
    else
    {
        typedef typename prnn::util::RemoveFirstType<Functions>::type RemainingFunctions;

        forwardPropRecurrentOverActivationFunctions(activations, weights, scratch, handle,
            RemainingFunctions());
    }
}

void forwardPropRecurrentOverActivationFunctions(const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    forwardPropRecurrentOverActivationFunctions(activations, weights, scratch,
        handle, prnn::matrix::AllRecurrentForwardOps());
}

void genericForwardPropRecurrent(
    const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const RecurrentOpsHandle& handle)
{
    bool reversed = (handle.direction == prnn::RECURRENT_REVERSE);

    size_t timesteps     = activations.size()[2];
    size_t miniBatchSize = activations.size()[1];
    size_t layerSize     = activations.size()[0];

    size_t currentTimestep = reversed ? timesteps - 1 : 0;

    // Start value
    auto currentInput = slice(activations, {0, 0, currentTimestep},
        {layerSize, miniBatchSize, currentTimestep + 1});

    apply(currentInput, currentInput, *handle.activationFunction.forwardOperation);

    // Propagate through time
    for(size_t timestep = 1; timestep < timesteps; ++timestep)
    {
        currentTimestep = reversed ? timesteps - timestep - 1 : timestep;

        auto nextInput = slice(activations, {0, 0, currentTimestep},
            {layerSize, miniBatchSize, currentTimestep + 1});

        auto reshapedNextInput    = reshape(nextInput,    {layerSize, miniBatchSize});
        auto reshapedCurrentInput = reshape(currentInput, {layerSize, miniBatchSize});

        gemm(
            reshapedNextInput,           1.0,
            weights,              false, 1.0,
            reshapedCurrentInput, false);

        currentInput = nextInput;

        apply(currentInput, currentInput, *handle.activationFunction.forwardOperation);
    }

}

}

void forwardPropRecurrent(
    const matrix::DynamicView& activations,
    const matrix::ConstDynamicView& weights,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    if(!parallel::isCudaEnabled())
    {
        detail::genericForwardPropRecurrent(activations, weights, handle);
        return;
    }

    assert(activations.precision() == weights.precision());
    assert(activations.precision() == scratch.precision());

    zeros(scratch);

    detail::forwardPropRecurrentOverActivationFunctions(activations, weights, scratch, handle);
}

namespace detail
{

template <typename ActivationFunction, typename ArchitectureConfig>
void dispatchBackPropDeltasRecurrent(
    typename ArchitectureConfig::RealType* deltas,
    const typename ArchitectureConfig::RealType* weights,
    typename ArchitectureConfig::RealType* activations,
    typename ArchitectureConfig::RealType* scratch, const ArchitectureConfig& archParameters)
{
    typedef typename ArchitectureConfig::RealType RealType;

    size_t activationCount = archParameters.handle.layerSize;
    size_t miniBatchSize   = archParameters.handle.miniBatchSize;
    size_t timesteps       = archParameters.handle.timesteps;

    util::log("RecurrentOperations") << "Launch back propagation with "
        << archParameters.block_count() << " blocks ("
        << archParameters.thread_count() << " threads), each handling "
        << archParameters.activations_per_block() << " activations out of "
        << activationCount << " total, mini batch size " << miniBatchSize << ", timesteps "
        << timesteps << ".\n";

    hipStream_t stream = reinterpret_cast<hipStream_t>(archParameters.handle.stream);

    Synchronizer synchronizer(archParameters.block_count(), stream,
        getSynchronizerScratch(scratch, archParameters));

    typedef typename ArchitectureConfig::TileParameters TileConfig;

    typedef RecurrentConfig<RealType, ActivationFunction, TileConfig> Config;

    Config config(archParameters.handle);

    while(synchronizer.not_finished()) {
        PersistentEngineParameters<Config> parameters(config, weights, activations,
            deltas, scratch, archParameters.handle.skipConnectionScale, synchronizer);

        back_prop_recurrent_deltas_kernel<<<archParameters.blocks(),
            archParameters.threads(), 0, stream>>>(parameters);

        synchronizer.check_for_failure();

        if (synchronizer.not_finished()) {
            util::log("RecurrentOperations") << " back prop launch failed, restarting at phase "
                << synchronizer.get_current_phase() << ".\n";
            synchronizer.reset_failed_flag();
        }
    }

}

template <typename ActivationFunction, typename T, RecurrentLayerDirection direction>
void backPropDeltasRecurrent(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<T>& precision)
{
    typedef typename T::type RealType;

    const RealType* weightsData    = weights.data<RealType>();
          RealType* activationData = activations.data<RealType>();
          RealType* deltaData      = deltas.data<RealType>();
          RealType* scratchData    = scratch.data<RealType>();

    int major = 0;
    int minor = 0;
    int smCount = 0;

    getGPUMajorAndMinorVersion(major, minor, smCount);

    if(major == 6 && smCount >= 60)
    {
        typedef typename TileSelector<direction, RealType, 60, 6>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchBackPropDeltasRecurrent<ActivationFunction, ArchParams>(deltaData,
            weightsData, activationData, scratchData, architectureConfig);
    }
    else if(major == 5 && smCount >= 24)
    {
        typedef typename TileSelector<direction, RealType, 24, 5>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchBackPropDeltasRecurrent<ActivationFunction, ArchParams>(deltaData,
            weightsData, activationData, scratchData, architectureConfig);
    }
    else
    {
        typedef typename TileSelector<direction, RealType, 1, 0>::TileSize TileSize;
        typedef RecurrentArchitectureParameters<RealType, TileSize> ArchParams;

        ArchParams architectureConfig(handle);

        dispatchBackPropDeltasRecurrent<ActivationFunction, ArchParams>(deltaData,
            weightsData, activationData, scratchData, architectureConfig);
    }
}

template <typename ActivationFunction, typename T>
void backPropDeltasRecurrentOverPrecisions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<T>& precision)
{
    typedef T PossiblePrecision;

    assert(PossiblePrecision() == activations.precision());

    if(handle.direction == prnn::RECURRENT_REVERSE)
    {
        backPropDeltasRecurrent<ActivationFunction, PossiblePrecision, prnn::RECURRENT_FORWARD>(
            deltas, weights, activations, scratch, handle, precision);
    }
    else
    {
        backPropDeltasRecurrent<ActivationFunction, PossiblePrecision, prnn::RECURRENT_REVERSE>(
            deltas, weights, activations, scratch, handle, precision);
    }
}

template<typename ActivationFunction, typename Precisions>
void backPropDeltasRecurrentOverPrecisions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const Precisions& precisions)
{
    typedef typename std::tuple_element<0, Precisions>::type PossiblePrecision;

    if(activations.precision() == PossiblePrecision())
    {
        backPropDeltasRecurrentOverPrecisions<ActivationFunction>(
            deltas, weights, activations, scratch, handle, std::tuple<PossiblePrecision>());
    }
    else
    {
        typedef typename util::RemoveFirstType<Precisions>::type RemainingPrecisions;

        backPropDeltasRecurrentOverPrecisions<ActivationFunction>(deltas, weights, activations,
            scratch, handle, RemainingPrecisions());
    }
}

template <typename ActivationFunction>
void backPropDeltasRecurrentOverPrecisions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    backPropDeltasRecurrentOverPrecisions<ActivationFunction>(deltas, weights, activations,
        scratch, handle, prnn::matrix::AllPrecisions());
}

template<typename ActivationFunction>
void backPropDeltasRecurrentOverActivationFunctions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const std::tuple<ActivationFunction>& activationFunction)
{
    assert(ActivationFunction() == *handle.activationFunction.reverseOperation);

    backPropDeltasRecurrentOverPrecisions<ActivationFunction>(deltas,
        weights, activations, scratch, handle);
}

template<typename Functions>
void backPropDeltasRecurrentOverActivationFunctions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle,
    const Functions& functions)
{
    typedef typename std::tuple_element<0, Functions>::type PossibleFunction;

    if(*handle.activationFunction.reverseOperation == PossibleFunction())
    {
        backPropDeltasRecurrentOverActivationFunctions(deltas, weights, activations,
            scratch, handle, std::tuple<PossibleFunction>());
    }
    else
    {
        typedef typename prnn::util::RemoveFirstType<Functions>::type RemainingFunctions;

        backPropDeltasRecurrentOverActivationFunctions(deltas, weights, activations,
            scratch, handle, RemainingFunctions());
    }
}

void backPropDeltasRecurrentOverActivationFunctions(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    backPropDeltasRecurrentOverActivationFunctions(deltas, weights, activations, scratch,
        handle, prnn::matrix::AllRecurrentBackwardOps());
}

void genericBackPropDeltasRecurrent(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const RecurrentOpsHandle& handle)
{
    bool reversed = (handle.direction == prnn::RECURRENT_REVERSE);

    size_t maxTimesteps  = deltas.size()[2];
    size_t miniBatchSize = deltas.size()[1];
    size_t layerSize     = deltas.size()[0];

    auto currentTimestep = reversed ? 0 : maxTimesteps - 1;

    // Start value
    auto currentDeltas = slice(deltas,
        {0, 0, currentTimestep}, {layerSize, miniBatchSize, currentTimestep + 1});
    auto currentActivations = slice(activations,
        {0, 0, currentTimestep}, {layerSize, miniBatchSize, currentTimestep + 1});

    apply(currentDeltas, currentActivations, currentDeltas,
        *handle.activationFunction.reverseOperation);

    // go over all timesteps in reverse
    for(size_t t = 1; t < maxTimesteps; ++t)
    {
        size_t timestep = reversed ? t : maxTimesteps - t - 1;

        auto previousDeltas = slice(deltas, {0, 0, timestep},
            {layerSize, miniBatchSize, timestep + 1});

        auto reshapedPreviousDeltas = reshape(previousDeltas, {layerSize, miniBatchSize});
        auto reshapedCurrentDeltas  = reshape(currentDeltas,  {layerSize, miniBatchSize});

        gemm(
            reshapedPreviousDeltas, 1.0,
            weights, true, 1.0,
            reshapedCurrentDeltas, false
        );

        currentDeltas = previousDeltas;

        currentActivations = slice(activations, {0, 0, timestep},
            {layerSize, miniBatchSize, timestep + 1});

        apply(currentDeltas, currentActivations, currentDeltas,
            *handle.activationFunction.reverseOperation);
    }

}

}

void backPropDeltasRecurrent(const matrix::DynamicView& deltas,
    const matrix::ConstDynamicView& weights, const matrix::DynamicView& activations,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    if(!parallel::isCudaEnabled())
    {
        detail::genericBackPropDeltasRecurrent(deltas, weights, activations, handle);
        return;
    }

    zeros(scratch);

    detail::backPropDeltasRecurrentOverActivationFunctions(deltas, weights, activations,
        scratch, handle);
}

void backPropGradientsRecurrent(const matrix::DynamicView& dWeights,
    const matrix::ConstDynamicView& outputActivations,
    const matrix::ConstDynamicView& deltas,
    const matrix::DynamicView& scratch, const RecurrentOpsHandle& handle)
{
    bool reversed = (handle.direction == prnn::RECURRENT_REVERSE);

    size_t timesteps     = handle.timesteps;
    size_t miniBatchSize = handle.miniBatchSize;
    size_t layerSize     = handle.layerSize;

    // Compute gradients
    size_t start = reversed ? 0 : 1;

    auto slicedDeltas = slice(deltas,
                             {0,         0,             start},
                             {layerSize, miniBatchSize, timesteps - (1 - start)});


    auto slicedActivations = slice(outputActivations,
                                  {0,         0,             1         - start},
                                  {layerSize, miniBatchSize, timesteps - start});

    gemm(dWeights, 0.0,
         reshape(slicedDeltas,
                {layerSize, miniBatchSize * (timesteps - 1)}), false, 1.0,
         reshape(slicedActivations,
                {layerSize, miniBatchSize * (timesteps - 1)}), true);

}

static matrix::Dimension extendDimensions(const matrix::Dimension& dimensions,
    const matrix::Precision& precision)
{
    auto newDimensions = dimensions;

    newDimensions[0] = prnn::rnn::getMaximumSizeRNNForThisGPU(precision);
    newDimensions[2] += 1;

    return newDimensions;
}

matrix::Matrix getForwardPropScratch(const RecurrentOpsHandle& handle,
    const matrix::Precision& precision)
{
    matrix::Dimension dimension(handle.layerSize, handle.miniBatchSize, handle.timesteps);

    auto scratchDimension = extendDimensions(dimension, precision);

    return matrix::Matrix(scratchDimension, precision);
}


matrix::Matrix getBackPropDeltasScratch(const RecurrentOpsHandle& handle,
    const matrix::Precision& precision)
{
    return getForwardPropScratch(handle, precision);
}

matrix::Matrix getBackPropGradientsScratch(const RecurrentOpsHandle& handle,
    const matrix::Precision& precision)
{
    return matrix::Matrix();
}

}
}

